
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

#define ni 25088
#define nn 4096

#define ti 32
#define tn 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; i++) 
   {
      a[i] = rand(); 
   }
}

void zeros(int* a, int N)
{
   int i;
   for (i = 0; i < N; i++) 
   {
      a[i] = 0; 
   }
}

// CURRENT MEMORY PERFORMANCE = 20.85 MB/s

// perform 1 tile of the matrix-vector multiply (subset of input, subset of weights matrix)
// this means that the batch size is (ti*tn)(?)
// the dimensions of the weights matrix are (ni, nn) => 2D array
// the full input is a vector of dimension ni (represented as an array)
// the full output is a vector of dimension nn (represented as an array)
// this is what is done in a fully-connected classifier layer
// this method utilizes a scratchpad memory for better thread block performance
__global__
void matrix_vector_mult(int *inp, int *outp, int *kern)
{
   // scratchpad memory used for shared variables
   __shared__ int temp_inp[ti]; // partial input vector
   __shared__ int temp_kern[ti * tn]; // partial kernel matrix

   // populate shared data structures
   int help = (blockIdx.x % (ni/ti));
   if (threadIdx.x % ti == 0) {
      int hold = threadIdx.x / ti;
      int i_index = help + hold;
      temp_inp[hold] = inp[i_index];
   }

   int k_index = (help * (tn + ti) + (threadIdx.x % ti) * ti + (threadIdx.x % tn)); // 1) get to first element of correct submatrix 2) get to desired row of submatrix 3) get to desired element of submatrix
   temp_kern[threadIdx.x] = kern[k_index]; 
   
   __syncthreads(); // sync all threads to this point 

   // populate output
   int n_index = (blockIdx.x % (nn/tn)) + (threadIdx.x % ti);
   outp[n_index] += temp_kern[threadIdx.x] * temp_inp[threadIdx.x % ti];
}

int main(void)
{
    // declare host + device pointers
    int *inp, *outp, *kern;
    int *d_inp, *d_outp, *d_kern;
    
    // compute array sizes
    int i_size = ni;
    int o_size = nn;
    int k_size = nn*ni;
    
    // allocate space for each array on the device
    gpuErrchk( hipMalloc(&d_inp, i_size*sizeof(int)) );
    gpuErrchk( hipMalloc(&d_outp, o_size*sizeof(int)) );
    gpuErrchk( hipMalloc(&d_kern, k_size*sizeof(int)) );
    
    // allocate space and populate each array on the host
    inp = (int*)malloc(i_size*sizeof(int)); 
    outp = (int*)malloc(o_size*sizeof(int));
    kern = (int*)malloc(k_size*sizeof(int)); 
    random_ints(inp, i_size);
    zeros(outp, o_size);
    random_ints(kern, k_size);
    
    // copy populated host arrays to corresponding device arrays
    gpuErrchk( hipMemcpy(d_inp, inp, i_size*sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_outp, outp, o_size*sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_kern, kern, k_size*sizeof(int), hipMemcpyHostToDevice) );

    // launch all threads on device
    // # blocks = # submatrices (tiles)
    // # threads / block = # elements per submatrix (tile)
    matrix_vector_mult<<<(ni*nn)/(ti*tn), ti*tn>>>(d_inp, d_outp, d_kern);
    
    // determine if run succeeded
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    
    // copy output array back to host
    gpuErrchk( hipMemcpy(outp, d_outp, o_size, hipMemcpyDeviceToHost) );

    // free all memory
    free(inp); free(outp); free(kern);
    gpuErrchk( hipFree(d_inp) ); gpuErrchk( hipFree(d_outp) ); gpuErrchk( hipFree(d_kern) );

    return 0;
}